#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <cfloat>

#include "gpuspatial/geom/id_encoder.cuh"
#include "gpuspatial/geom/line_segment.cuh"
#include "gpuspatial/index/detail/launch_parameters.h"
#include "gpuspatial/utils/floating_point.h"
#include "shader_config.h"

enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };
// FLOAT_TYPE is defined by CMakeLists.txt
extern "C" __constant__ gpuspatial::detail::LaunchParamsMultiPolygonPointQuery<
    gpuspatial::ShaderPointType, uint32_t>
    params;

extern "C" __global__ void __intersection__gpuspatial() {
  using namespace gpuspatial;
  using point_t = ShaderPointType;
  using scalar_t = typename point_t::scalar_t;
  auto aabb_id = optixGetPrimitiveIndex();
  auto query_idx = optixGetPayload_0();
  auto reordered_multi_polygon_idx = optixGetPayload_1();
  uint32_t v_offset = optixGetPayload_2();
  auto part_idx = optixGetPayload_3();
  auto ring_idx = optixGetPayload_4();
  auto crossing_count = optixGetPayload_5();
  auto point_on_seg = optixGetPayload_6();
  const auto& multi_polygons = params.multi_polygons;
  auto multi_polygon_idx = params.ids[query_idx].first;
  auto point_idx = params.ids[query_idx].second;

  // the seg being hit is not from the query polygon
  if (params.seg_multi_polygon_ids[aabb_id] != multi_polygon_idx) {
    return;
  }

  uint32_t local_v1_idx = aabb_id - params.seg_begins[reordered_multi_polygon_idx];
  uint32_t global_v1_idx = v_offset + local_v1_idx;
  uint32_t global_v2_idx = global_v1_idx + 1;
  uint32_t hit_geom_idx, hit_part_idx, hit_ring_idx;

  // bool found = multi_polygons.locate_vertex(global_v1_idx, hit_geom_idx, hit_part_idx,
                                            // hit_ring_idx);

  // assert(params.geom_ids[aabb_id] == hit_geom_idx);
  // assert(params.part_ids[aabb_id] == hit_part_idx);
  // assert(params.ring_ids[aabb_id] == hit_ring_idx);
  // assert(found);

  hit_geom_idx = params.geom_ids[aabb_id];
  hit_part_idx = params.part_ids[aabb_id];
  hit_ring_idx = params.ring_ids[aabb_id];

  if (hit_geom_idx == multi_polygon_idx && hit_part_idx == part_idx &&
      hit_ring_idx == ring_idx) {
    auto vertices = multi_polygons.get_vertices();
    // segment being hit
    const auto& v1 = vertices[global_v1_idx];
    const auto& v2 = vertices[global_v2_idx];
    const auto& p = params.points[point_idx];

    RayCrossingCounter locator(crossing_count, point_on_seg);
    locator.countSegment(p, v1, v2);
    optixSetPayload_5(locator.get_crossing_count());
    optixSetPayload_6(locator.get_point_on_segment());
  }
}

extern "C" __global__ void __raygen__gpuspatial() {
  using namespace gpuspatial;
  float tmin = 0;
  float tmax = FLT_MAX;  // use a very large value
  const auto& ids = params.ids;
  using point_t = ShaderPointType;
  const auto& multi_polygons = params.multi_polygons;
  RayCrossingCounter locator;

  for (uint32_t i = optixGetLaunchIndex().x; i < ids.size();
       i += optixGetLaunchDimensions().x) {
    auto multi_polygon_idx = ids[i].first;
    auto point_idx = ids[i].second;

    auto it = thrust::lower_bound(thrust::seq, params.multi_polygon_ids.begin(),
                                  params.multi_polygon_ids.end(), multi_polygon_idx);
    assert(it != params.multi_polygon_ids.end());
    uint32_t reordered_multi_polygon_idx =
        thrust::distance(params.multi_polygon_ids.begin(), it);
    assert(params.multi_polygon_ids[reordered_multi_polygon_idx] == multi_polygon_idx);

    const auto& p = params.points[point_idx];

    float3 origin;
    // each polygon takes a z-plane
    origin.x = p.x();
    origin.y = p.y();
    // origin.z = reordered_multi_polygon_idx;
    // cast ray toward positive x-axis
    float3 dir = {1, 0, 0};
    auto part_begin = params.part_begins[i];
    const auto& multi_polygon = multi_polygons[multi_polygon_idx];
    const auto& mbr = multi_polygon.get_mbr();
    auto width = mbr.get_max().x() - mbr.get_min().x();
    tmax = 2 * width;

    // first polygon offset
    uint32_t part_offset = multi_polygons.get_prefix_sum_geoms()[multi_polygon_idx];
    // first ring offset of the polygon
    uint32_t ring_offset = multi_polygons.get_prefix_sum_parts()[part_offset];
    // first vertex offset of the ring
    uint32_t v_offset = multi_polygons.get_prefix_sum_rings()[ring_offset];

    for (uint32_t part = 0; part < multi_polygon.num_polygons(); part++) {
      auto polygon = multi_polygon.get_polygon(part);
      uint32_t ring = 0;
      locator.Init();
      uint32_t encoded_z = ENCODE_UINT32_T_3(reordered_multi_polygon_idx, part, ring);
      origin.z = *reinterpret_cast<float*>(&encoded_z);
      uint32_t n_hits = 0;
      // test exterior
      optixTrace(params.handle, origin, dir, tmin, tmax, 0, OptixVisibilityMask(255),
                 OPTIX_RAY_FLAG_NONE,            // OPTIX_RAY_FLAG_NONE,
                 SURFACE_RAY_TYPE,               // SBT offset
                 RAY_TYPE_COUNT,                 // SBT stride
                 SURFACE_RAY_TYPE,               // missSBTIndex
                 i,                              // 0
                 reordered_multi_polygon_idx,    // 1
                 v_offset,                       // 2
                 part,                           // 3
                 ring,                           // 4
                 locator.get_crossing_count(),   // 5
                 locator.get_point_on_segment()  // 6
      );
      auto location = locator.location();
      PointLocation final_location = PointLocation::kError;
      if (location == PointLocation::kInside) {
        final_location = location;
        // test interior
        for (ring = 1; ring < polygon.num_rings(); ring++) {
          n_hits = 0;
          locator.Init();
          encoded_z = ENCODE_UINT32_T_3(reordered_multi_polygon_idx, part, ring);
          origin.z = *reinterpret_cast<float*>(&encoded_z);
          optixTrace(params.handle, origin, dir, tmin, tmax, 0, OptixVisibilityMask(255),
                     OPTIX_RAY_FLAG_NONE,            // OPTIX_RAY_FLAG_NONE,
                     SURFACE_RAY_TYPE,               // SBT offset
                     RAY_TYPE_COUNT,                 // SBT stride
                     SURFACE_RAY_TYPE,               // missSBTIndex
                     i,                              // 0
                     reordered_multi_polygon_idx,    // 1
                     v_offset,                       // 2
                     part,                           // 3
                     ring,                           // 4
                     locator.get_crossing_count(),   // 5
                     locator.get_point_on_segment()  // 6
          );
          location = locator.location();
          if (location == PointLocation::kBoundary) {
            final_location = PointLocation::kBoundary;
            break;
          } else if (location == PointLocation::kInside) {
            final_location = PointLocation::kOutside;
            break;
          }
        }
      } else {
        // outside or boundary
        final_location = location;
      }
      assert(final_location != PointLocation::kError);
      params.locations[part_begin + part] = final_location;
#ifndef NDEBUG
      auto ref_loc =
          multi_polygon.get_polygon(part).locate_point(params.points[point_idx]);
      if (ref_loc != final_location) {
        printf(
            "reorder %u, multi poly %u, point %u (%lf, %lf), num parts %u, num rings %u, part %u, point %u, loc %d, ref loc %d\n",
            reordered_multi_polygon_idx, multi_polygon_idx, point_idx, p.x(), p.y(),
            multi_polygon.num_polygons(), multi_polygon.get_polygon(0).num_rings(), part,
            point_idx, (int)final_location, (int)ref_loc);
        assert(false);
      }
#endif
    }
  }
}
