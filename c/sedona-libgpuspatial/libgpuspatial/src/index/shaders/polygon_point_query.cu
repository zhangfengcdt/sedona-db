#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>
#include <cfloat>

#include "gpuspatial/geom/line_segment.cuh"
#include "gpuspatial/index/detail/launch_parameters.h"
#include "shader_config.h"

enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };
// FLOAT_TYPE is defined by CMakeLists.txt
extern "C" __constant__ gpuspatial::detail::LaunchParamsPolygonPointQuery<
    gpuspatial::ShaderPointType, uint32_t>
    params;

extern "C" __global__ void __intersection__gpuspatial() {
  using namespace gpuspatial;
  using point_t = ShaderPointType;
  using equation_t = EdgeEquation<point_t>;
  auto aabb_id = optixGetPrimitiveIndex();
  auto query_idx = optixGetPayload_0();
  auto point_idx = optixGetPayload_1();
  auto polygon_idx = optixGetPayload_2();            // polygon id in the polygon array
  auto reordered_polygon_idx = optixGetPayload_3();  // ith polygon from "ids"

  if (params.seg_polygon_ids[aabb_id] != polygon_idx) {
    return;
  }

  auto local_v1_idx = aabb_id - params.seg_begins[reordered_polygon_idx];
  const auto& polygons = params.polygons;
  // first ring offset of the polygon
  auto ring_offset = polygons.get_prefix_sum_polygons()[polygon_idx];
  // first vertex offset of the ring
  auto v_offset = polygons.get_prefix_sum_rings()[ring_offset];
  auto global_v1_idx = v_offset + local_v1_idx;
  auto global_v2_idx = global_v1_idx + 1;
  uint32_t v1_polygon_idx, v1_ring_idx;
  bool found = polygons.locate_vertex(global_v1_idx, v1_polygon_idx, v1_ring_idx);
  assert(found);
  assert(v1_polygon_idx == polygon_idx);

  auto vertices = polygons.get_vertices();
  const auto& v1 = vertices[global_v1_idx];
  const auto& v2 = vertices[global_v2_idx];
  auto x_min = std::min(v1.x(), v2.x());
  auto x_max = std::max(v1.x(), v2.x());

  // query point
  const auto& p = params.points[point_idx];

  // out of range, the point cannot hit the seg
  if (p.x() < x_min || p.x() > x_max) {
    return;
  }


  // this is a vertical seg
  if (x_min == x_max) {
    auto y_min = std::min(v1.y(), v2.y());
    auto y_max = std::max(v1.y(), v2.y());
    // p is on the seg
    if (x_min == p.x() && p.y() >= y_min && p.y() <= y_max) {
      params.locations[query_idx] = PointLocation::kBoundary;
    }
    return;
  }

  equation_t e(v1, v2);
  // now, the seg should not be vertical
  assert(e.b != 0);

  auto xsect_y = (-e.a * p.x() - e.c) / e.b;

  if (e.a * p.x() + e.b * p.y() + e.c == 0) {
    params.locations[query_idx] = PointLocation::kBoundary;
  }

  // printf("hit, x min %lf, max %lf, x %lf, xsect y %lf, y %lf\n", x_min, x_max, p.x(),
    // xsect_y, p.y());


  auto diff_y = xsect_y - p.y();
  // current point is above the current edge
  if (diff_y < 0) {
    return;
  }

  uint2 best_y_storage{optixGetPayload_4(), optixGetPayload_5()};
  double best_y;
  unpack64(best_y_storage.x, best_y_storage.y, &best_y);

  // printf("xsect y %lf\n", xsect_y);

  // find closest line seg to the query point
  if (xsect_y < best_y) {
    PointLocation location = PointLocation::kError;
    // exterior ring is counterclockwise
    // interior ring is clockwise , so they have consistent faces
    PointLocation left_face = PointLocation::kInside;
    PointLocation right_face = PointLocation::kOutside;

    if (v1.x() < v2.x()) {
      location = right_face;
    } else {
      location = left_face;
    }
    // printf("location %d\n", (int)location);
    params.locations[query_idx] = location;
    best_y = xsect_y;
    pack64(&best_y, best_y_storage.x, best_y_storage.y);
    optixSetPayload_4(best_y_storage.x);
    optixSetPayload_5(best_y_storage.y);
    optixReportIntersection(diff_y, 0);
  }
}

extern "C" __global__ void __raygen__gpuspatial() {
  using namespace gpuspatial;
  float tmin = 0;
  float tmax = 1e10; // use a very large value
  const auto& ids = params.ids;
  for (uint32_t i = optixGetLaunchIndex().x; i < ids.size();
       i += optixGetLaunchDimensions().x) {
    auto polygon_idx = ids[i].first;
    auto point_idx = ids[i].second;

    // if (!(polygon_idx == 947 && point_idx == 486)) {
    //   return;
    // }


    // auto pip = params.polygons[polygon_idx].Contains(params.points[point_idx]);
    // if (pip) {
    //   printf("polygon %u, point %u\n", polygon_idx, point_idx);
    // }

    auto it = thrust::lower_bound(thrust::seq, params.polygon_ids.begin(),
                                  params.polygon_ids.end(), polygon_idx);
    assert(it != params.polygon_ids.end());
    uint32_t reordered_polygon_idx = thrust::distance(params.polygon_ids.begin(), it);
    assert(params.polygon_ids[reordered_polygon_idx] == polygon_idx);

    const auto& p = params.points[point_idx];

    float3 origin;
    // each polygon takes a z-plane
    origin.x = p.x();
    origin.y = p.y();
    origin.z = reordered_polygon_idx;
    // cast ray toward positive y-axis
    float3 dir = {0, 1, 0};

    auto best_y = std::numeric_limits<double>::max();
    // best means closest line seg from the query point
    uint2 best_y_storage;

    pack64(&best_y, best_y_storage.x, best_y_storage.y);

    // printf("ray origin (%lf, %lf), reorder %u\n", params.points[point_idx].x(), params.points[point_idx].y(), reordered_polygon_idx);


    optixTrace(params.handle, origin, dir, tmin, tmax, 0, OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,     // SBT offset
               RAY_TYPE_COUNT,       // SBT stride
               SURFACE_RAY_TYPE,     // missSBTIndex
               i, point_idx, polygon_idx, reordered_polygon_idx, best_y_storage.x,
               best_y_storage.y);
    unpack64(best_y_storage.x, best_y_storage.y, &best_y);
    // the ray hits nothing, so point is outside of the polygon
    if (best_y == std::numeric_limits<double>::max()) {
      params.locations[i] = PointLocation::kOutside;
    }

    auto loc = params.locations[i];
    auto ref_loc = params.polygons[polygon_idx].locate_point(params.points[point_idx]);
    // if (loc != ref_loc) {
    //   printf("poly %u, point %u, loc %d, ref loc %d\n", polygon_idx, point_idx, (int)loc,
    //          (int)ref_loc);
    // }
  }
}
